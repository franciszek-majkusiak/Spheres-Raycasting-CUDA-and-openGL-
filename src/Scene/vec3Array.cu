#include "hip/hip_runtime.h"
#include "vec3Array.h"
#include <algorithm>
void InitCPU(vec3Arr& arr, int n)
{
	//printf("Init vec3Arr CPU\n");
	float* x;
	float* y;
	float* z;
	x = (float*)malloc(n * sizeof(float));
	y = (float*)malloc(n * sizeof(float));
	z = (float*)malloc(n * sizeof(float));

	arr.x = x;
	arr.y = y;
	arr.z = z;
}

void FreeCPU(vec3Arr& arr)
{
	//printf("Free vec3Arr CPU\n");
	free(arr.x);
	free(arr.y);
	free(arr.z);
}

void Swap(vec3Arr& arr1, vec3Arr& arr2)
{
	std::swap(arr1.x, arr2.x);
	std::swap(arr1.y, arr2.y);
	std::swap(arr1.z, arr2.z);
}



void Init(vec3Arr& arr, int n)
{
	//printf("Init GPU vec3Arr\n");
	float* x = 0;
	float* y = 0;
	float* z = 0;
	size_t size = n * sizeof(float);
	checkCudaErrors(hipMalloc((void**)&x, size));
	checkCudaErrors(hipMalloc((void**)&y, size));
	checkCudaErrors(hipMalloc((void**)&z, size));

	arr.x = x;
	arr.y = y;
	arr.z = z;
}

void Free(vec3Arr& arr)
{
	//printf("Free GPU vec3Arr\n");
	hipFree(arr.x);
	hipFree(arr.y);
	hipFree(arr.z);
}

void CopyHostToDevice(vec3Arr& d_arr, vec3Arr& h_arr, int n)
{
	checkCudaErrors(hipMemcpy(d_arr.x, h_arr.x, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_arr.y, h_arr.y, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_arr.z, h_arr.z, n * sizeof(float), hipMemcpyHostToDevice));
}

float RandomFloat(float min, float max)
{
	return (float)rand() / (float)RAND_MAX * (max - min) + min;
}